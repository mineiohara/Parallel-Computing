#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 100000000
#define Max 1024

__global__ void add1( int *a, int *b, int *c );
__global__ void add2( int *a, int *b, int *c );
__global__ void add3( int *a, int *b, int *c );

int main(){
    int *a, *b, *c;
    int *da, *db, *dc;
    int i;
    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate the memory on the CPU
    a = (int *)malloc(Max * sizeof(int));
    b = (int *)malloc(Max * sizeof(int));
    c = (int *)malloc(Max * sizeof(int));

    // Allocate the memory one the GPU
    hipMalloc((void **)&da, Max * sizeof(int));
    hipMalloc((void **)&db, Max * sizeof(int));
    hipMalloc((void **)&dc, Max * sizeof(int));

    srand(time(NULL));
    for (i = 0; i < Max; i++){
        a[i] = rand()%256;
        b[i] = rand()%256;
    }

    // Copy the arrays to GPU
    hipMemcpy(da, a, Max * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, Max * sizeof(int), hipMemcpyHostToDevice);

    // One block with multiple threads
    hipEventRecord(start, 0);
    add1<<<1, Max>>>(da, db, dc);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c, dc, Max * sizeof(int), hipMemcpyDeviceToHost);
    for (i = 0; i < Max; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
        }
    }
    printf("One block with multiple threads: %13f msec\n", elapsedTime);

    // Multiple blocks, each block has one thread
    hipEventRecord(start, 0);
    add2<<<Max, 1>>>(da, db, dc);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c, dc, Max * sizeof(int), hipMemcpyDeviceToHost);
    for (i = 0; i < Max; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
        }
    }
    printf("Multiple blocks/one thread: %13f msec\n", elapsedTime);


    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);

    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));

    hipMalloc((void **)&da, N * sizeof(int));
    hipMalloc((void **)&db, N * sizeof(int));
    hipMalloc((void **)&dc, N * sizeof(int));

    srand(time(NULL));
    for (i = 0; i < N; i++){
        a[i] = rand()%256;
        b[i] = rand()%256;
    }

    // Copy the arrays to GPU
    hipMemcpy(da, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, N * sizeof(int), hipMemcpyHostToDevice);


    // 1024 threads with N+1024-1/1024 blocks
    hipEventRecord(start, 0);
    add3<<<(N+Max-1)/Max, Max>>>(da, db, dc);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c, dc, N * sizeof(int), hipMemcpyDeviceToHost);
    for (i = 0; i < N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
        }
    }
    printf("1024 threads per block: %13f msec\n", elapsedTime);


    // 544 threads with N+544-1/544 blocks
    hipEventRecord(start, 0);
    add3<<<(N+544-1)/544, 544>>>(da, db, dc);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c, dc, N * sizeof(int), hipMemcpyDeviceToHost);
    for (i = 0; i < N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
        }
    }
    printf("544 threads per block: %13f msec\n", elapsedTime);


    // 32 threads with N+32-1/32 blocks
    hipEventRecord(start, 0);
    add3<<<(N+32-1)/32, 32>>>(da, db, dc);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(c, dc, N * sizeof(int), hipMemcpyDeviceToHost);
    for (i = 0; i < N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
        }
    }
    printf("32 threads per block: %13f msec\n", elapsedTime);
    

    // Free the memory allocated on the GPU
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

__global__ void add1(int *a, int *b, int *c){
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

__global__ void add2(int *a, int *b, int *c){
    int tid = blockIdx.x;
    c[tid] = a[tid] + b[tid];
}

__global__ void add3(int *a, int *b, int *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    c[tid] = a[tid] + b[tid];
}